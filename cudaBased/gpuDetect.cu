
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
// #include <typeinfo>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
//#include <nvml.h> // If I use this I'll need to start a context first.

// "MV2_COMM_WORLD_LOCAL_RANKs"
// #define ENV_LOCAL_RANKs		"OMPI_COMM_WORLD_LOCAL_RANKs"

int ranks, nthreads, nGpu, rlen=800;
hipDeviceProp_t props;


using namespace std;

void getDeviceInformation()
{
    char procN[rlen];
    hipGetDeviceCount(&nGpu);
    MPI_Get_processor_name(procN,  &rlen);

    cout << "---------- " << procN << " --------" << endl;

    if (nGpu)
    {
        for (int k=0; k<nGpu; k++)
        {
            hipGetDeviceProperties(&props, k);
            cout << ranks << " " << procN << " " << nthreads << " " << nGpu;
            cout << " " << k << " "<< props.name << " " << props.multiProcessorCount;
            cout << " " << props.major << "." << props.minor << endl;
        }
    }
    else
    {
        cout << ranks << " " << procN << " " << nthreads << " " << nGpu << endl;
    }

    //Maybe just set cuda compute mode = 1 (exclusive)
    

    // From this I want what GPUs each proc can see, and how many threads they can make
    // This may require nvml to get the UUID of the GPUS, pass them all up to the 
    // Master proc to decide which proc gets which gpu.
}

// Test device sight.
int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &ranks);
    if (!ranks) cout << "ranks -- cpuname -- nThreads -- nGpu -- GPU# -- GPUname -- #SM -- CC" << endl;
    getDeviceInformation();
    MPI_Barrier(MPI_COMM_WORLD);
	MPI_Finalize();
}
