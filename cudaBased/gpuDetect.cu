
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <utility>  
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
//#include <nvml.h> // If I use this I'll need to start a context first.

// "MV2_COMM_WORLD_LOCAL_RANKs"
// #define ENV_LOCAL_RANKs		"OMPI_COMM_WORLD_LOCAL_RANKs"
#define RLEN 800

int ranks, nprocs, nGpu;
hipDeviceProp_t props;

using namespace std;

struct assigner {
    int bus, dom;
    char procN[RLEN];
    // void assigner(assigner &orig){
    //     this->bus = orig.bus;
    //     this->dom = orig.dom;
    //     this->procN = orig.procN;
    //    }
};

int getDeviceInformation(int *hG, assigner *pr, int cnt)
{
    assigner voy;
    cout << cnt << endl;
    int rb = RLEN;
    hipGetDeviceCount(&nGpu);
    MPI_Get_processor_name(voy.procN,  &rb);
    int domy, busy, tag=-1, hit=0;

    cout << nGpu << endl;
    cout << ranks << "---------- " << voy.procN << " --------" << endl;
    if (!cnt) hit = 1;

    for (int k=0; k<nGpu; k++)
    {
        hipGetDeviceProperties(&props, k);
        
        domy = props.pciDomainID;
        busy = props.pciBusID;
        for (int a=0; a<cnt; a++)
        {
            cout << props.name << endl;
            cout << voy.procN << " " << domy << endl;
            cout << pr[a].procN << " " << pr[a].dom << endl;
            
            if (!strcmp(pr[a].procN, voy.procN) && pr[a].dom == domy && pr[a].bus==busy)
            {
                cout << voy.procN << " " << busy << endl;  
                break;  // This GPU is taken.
            }
            cout << a << " " << cnt << endl;
            if (a == (cnt-1))
            {
                hit = 1;
            }
        }
        if (hit)
        {
            tag = k;
            break;
        }
    }

    if (hit)
    { 
        *hG = tag;
        cout << ranks << "  YES GPU!" << endl;
        cout << " " << tag << " "<< props.name << " " << props.multiProcessorCount;
        cout << " " << props.major << "." << props.minor << " " << props.pciDomainID << " " << props.pciBusID << " " << props.pciDeviceID << endl;
        voy.dom = domy;
        voy.bus = busy;
        pr[cnt] = voy;
        return ++cnt;
    }
    else
    {   
        *hG = -1;
        cout << ranks << "  NO GPU!" << endl;
        cout << ranks << " " << voy.procN << " " << endl;
    }
    return cnt;



    //Maybe just set cuda compute mode = 1 (exclusive)
    

    // From this I want what GPUs each proc can see, and how many threads they can make
    // This may require nvml to get the UUID of the GPUS, pass them all up to the 
    // Master proc to decide which proc gets which gpu.
}

// Test device sight.
int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &ranks);
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    //MPI_Status stat;
    MPI_Datatype atype;
    // assigner inuit;
    // inuit.dom = 1111;
    // inuit.bus = 2222;
    // strcpy(inuit.procN, "Pennsylvania");
    assigner ids[nprocs];
    MPI_Datatype typs[] = {MPI_INT, MPI_INT, MPI_CHAR};
    int nm[] = {1, 1, RLEN};
    MPI_Aint disp[] = {0, 4, 8};
    MPI_Type_create_struct(3, nm, disp, typs, &atype);
    MPI_Type_commit(&atype);
    int idz=0;
    int hasGpu;
    if (!ranks) cout << "ranks -- cpuname -- nGpu -- GPU# -- GPUname -- #SM -- CC" << endl;
    for (int k=0; k<nprocs; k++)
    {
        if(ranks == k) 
        {
            idz = getDeviceInformation(&hasGpu, &ids[0], idz);
        }
        MPI_Bcast(&idz, 1, MPI_INT, k, MPI_COMM_WORLD);
        MPI_Bcast(&ids, nprocs, atype, k, MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
    }

    MPI_Type_free(&atype);
    MPI_Barrier(MPI_COMM_WORLD);
	MPI_Finalize();
}
