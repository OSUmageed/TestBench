
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <utility>  
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <unistd.h>
#include <sstream>
//#include <nvml.h> // If I use this I'll need to start a context first.

// "MV2_COMM_WORLD_LOCAL_RANKs"
// #define ENV_LOCAL_RANKs		"OMPI_COMM_WORLD_LOCAL_RANKs"
// mpicc -o -lcuda -lcudart
#define RLEN 80

int ranks, nprocs, nGpu;

using namespace std;

struct hname{
    int ng;
    char hostname[RLEN];
};

typedef vector<hname> hvec;

int getHost(hvec &ids, hname *newHost)
{
    char machineName[RLEN];
    int rb = RLEN;
    int nGpu;
    hipGetDeviceCount(&nGpu);
    MPI_Get_processor_name(&machineName[0],  &rb);
    for (int i=0; i<ids.size(); i++)
    {
        if (!strcmp(ids[i].hostname, machineName))
        {
            return i;
        }
    }

    strcpy(newHost->hostname, machineName);
    newHost->ng = nGpu;
    return ids.size();
}

// Test device sight.
int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &ranks);
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
    //MPI_Status stat;
    
    hvec ledger;
    int machineID;
    int hasgpu = 0;

	hname hBuf;
    MPI_Datatype atype;
    MPI_Datatype typs[] = {MPI_INT, MPI_CHAR};
    int nm[] = {1, RLEN};
    MPI_Aint disp[] = {0, 4};
    MPI_Type_create_struct(2, nm, disp, typs, &atype);
    MPI_Type_commit(&atype);

	cout << "Before First Loop " << ranks << " " << ledger.size() << endl; 

    for (int k=0; k<nprocs; k++)
    {
        if(ranks == k) 
        {
            machineID = getHost(ledger, &hBuf);
        }
		
        MPI_Bcast(&hBuf, 1, atype, k, MPI_COMM_WORLD);
		if (ledger.size() > 0)
		{
			if (strcmp(hBuf.hostname, ledger.back().hostname))
			{
				ledger.push_back(hBuf);
			}
		}
		else
		{
			ledger.push_back(hBuf);
		}
    }

	if(!ranks) cout << "After first loop but before Split" << endl;
    
	MPI_Comm machineComm;
    MPI_Comm_split(MPI_COMM_WORLD, machineID, ranks, &machineComm);
    int machineRank, machineSize;
    MPI_Comm_rank(machineComm, &machineRank);
    MPI_Comm_size(machineComm, &machineSize);

    MPI_Barrier(MPI_COMM_WORLD);
    if(!ranks) cout << "After Split" << endl;

	int nGpu = ledger[machineID].ng;
    int pcivec[nGpu*3];
	hipDeviceProp_t props;

	if (machineRank == 0)
	{
		for (auto i: ledger) {
			cout << "Rank - host - gpus " << ranks << " " << i.hostname << " " << i.ng  << endl;
		}
	}
	MPI_Barrier(MPI_COMM_WORLD);
    
	if (machineRank == 0)
    {
		cout << nGpu << " " << ledger.size() << endl;
        for (int k = 0; k < nGpu; k++) 
        {    
            hipGetDeviceProperties(&props, k);
			cout << "Rank " << ranks << " device- " << k << " ";
			cout << props.name << " " << props.pciBusID << endl;

			pcivec[3*k] = props.pciDomainID;
			pcivec[3*k+1] = props.pciBusID;
			pcivec[3*k+2] = props.pciDeviceID;
        }        
    }

	MPI_Bcast(&pcivec[0], 3*nGpu, MPI_INT, 0, machineComm);
	MPI_Barrier(MPI_COMM_WORLD);
    
	if(!ranks)	cout << "After PCI Broadcast" << endl;
	int nset = 0;
    int dev;
	string pcistr;
	stringstream bufs;
	
// DEBUGGED TO HERE!
    
    for (int i = 1; i<machineSize; i++)
    {
        if ((nGpu - nset) == 0) 
        {
            break;
        }
        if (i == machineRank)
        {
			bufs << std::hex << pcivec[3*nset] << ":" <<  pcivec[3*nset+1] << ":" <<  pcivec[3*nset+2];
            cout << i << " " << bufs.str() << endl;
            hipDeviceGetByPCIBusId(&dev, bufs.str().c_str());
            hipSetDevice(dev);
            hipGetDeviceProperties(&props, dev);
            cout << "----------------------" << endl;
            cout << "Global Rank: " << ranks << " Machine Rank: " << machineRank << std::endl;
            cout << "On machine " << ledger[machineID].hostname << std::endl;
            cout << "Acquired GPU: " << props.name << " with pciID: " << bufs.str() << endl;
			hasgpu = 1;
            nset++;
        }
        MPI_Bcast(&nset, 1, MPI_INT, i, machineComm);
        MPI_Barrier(machineComm);
    }
    MPI_Barrier(MPI_COMM_WORLD);
	for(int k=0; k<nprocs; k++)
	{
		if (ranks == k)
		{
			cout << ranks << " " << nset << " " << machineRank;
			cout << " " << hasgpu << endl;
		}
		MPI_Barrier(MPI_COMM_WORLD);
		sleep(1);
	}
	
    MPI_Type_free(&atype);
    MPI_Comm_free(&machineComm);
    MPI_Barrier(MPI_COMM_WORLD);
	MPI_Finalize();
	return 0;
}
