#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <iostream>
#include <fstream>
#include <ostream>
#include <istream>

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <string>
#include <vector>

// nvcc ptrs.cu -o ./bin/ptrs -gencode arch=compute_35,code=sm_35 -lm -O3 -std=c++11

using namespace std;

__device__ int *a, *b; 

__global__ void setptr(int *sj, int offs, int *ps)
{
	ps = &sj[offs];
}


__global__ void someMth(int *sj)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x; //Global Thread ID
	int om = sj[gid+1];
	sj[gid] += om;
}

int main()
{
	int *fptr, *sptr;
	int mysize = 256;
	int realsz = mysize/2;
	int tpb = 32;
	int bks = realsz/tpb;	
	int rlsz = sizeof(int)*realsz;
	int imem[mysize];
	int rmem[mysize]; 
	int gmem[mysize]; 

	hipMalloc((void **) &fptr, sizeof(int)*mysize);
	for (int k=0; k<mysize; k++) imem[k] = k;

	hipMemcpy(fptr, &imem, sizeof(int) * mysize, hipMemcpyHostToDevice);

	someMth <<< bks, tpb >>> (fptr);

    hipMemcpy(&imem, fptr, rlsz, hipMemcpyDeviceToHost);

//	for (int k=0; k<realsz; k++) cout << imem[k] << " ";
//	cout << endl;

	setptr <<< 1,1 >>> (fptr, tpb, sptr); 

	someMth <<< bks, tpb  >>> (fptr+tpb);

	hipMemcpy(&rmem, fptr, rlsz+sizeof(int)*tpb, hipMemcpyDeviceToHost);
	hipMemcpy(&gmem, fptr + (tpb*2), rlsz+sizeof(int)*tpb, hipMemcpyDeviceToHost);

	cout << "IDx || After 1 || Swap" << endl;
	for (int k=0; k<realsz + tpb; k++) cout << k << " " << imem[k] << " " << rmem[k] << " " << gmem[k] << endl;

	return 0;
}
