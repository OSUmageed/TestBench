#include "hip/hip_runtime.h"
/** 
    Copyright (C) 2017 Kyle Niemeyer, niemeyek@oregonstate.edu AND
    Daniel Magee, mageed@oregonstate.edu
*/
/*
    This file is distribued under the MIT License.  See LICENSE at top level of directory or: <https://opensource.org/licenses/MIT>.
*/

#include "kernel.h"
//#include "coopKernel.h"

__global__ 
void classicStep(states *state, int ts)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int idxes[3];
    for (int k=-1; k<2; k++) idxes[k+1] = ((gid+k) & deqConstants.lastIndex); 
    
    stepUpdate(state, idxes, ts);
}

__device__
__forceinline__
void sweepRead(states *tState, states *statein, int gid, int tid, int bd)
{
    int tadj = tid * (bd + 1);
    tState[tid+1] = statein[gid];
    __syncthreads();
    if (tid<2)
    {
        if (gid == 0)
        {
            tState[0] = statein[deqConstants.lastIndex];
        }   
        else if (gid == deqConstants.lastIndex)
        {
            tState[blockDim.x + 1] = statein[0];
        }
        else
        {
            tState[tadj] = statein[(gid-1) + tadj];
        }
    }
}

__global__
void upTriangle(states *statein, states *stateout, int tstep)
{   
    extern __shared__ states tState[];

    //Global Thread ID
    int tid = threadIdx.x; // Thread index
    int gid = blockDim.x * blockIdx.x + threadIdx.x; 
    int tidx = threadIdx.x; //Block Thread ID
    int mid = blockDim.x >> 1;
    int gidout = (gid - mid) % deqConstants.lastIndex;
    int tnow = tstep;
    int idxes[3];
    for (int k=-1; k<2; k++) idxes[k+1] = tid + k;

    tState[tidx] = statein[gid];

    __syncthreads();

    for (int k=1; k<mid; k++)
    {
        if (tidx < (blockDim.x-k) && tidx >= k)
        {
            stepUpdate(tState, idxes, tnow); 
        }
        tnow++;
        __syncthreads();
    }
    stateout[gidout] = tState[tidx];
}

__global__
void downTriangle(states *statein, states *stateout, int tstep)
{
    extern __shared__ states tState[];

    int tid = threadIdx.x; // Thread index
    int mid = blockDim.x >> 1; // Half of block size
    int base = blockDim.x + 2; 
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int tidx = tid + 1;
    int tnow = tstep; // read tstep into register.
    int idxes[3];
    for (int k=-1; k<2; k++) idxes[k+1] = tidx + k;

    sweepRead(tState, statein, gid, tid, blockDim.x);
    
    for (int k=mid; k>0; k--)
    {
        if (tidx < (base-k) && tidx >= k)
        {
                stepUpdate(tState, idxes, tnow);
        }
        tnow++;
        __syncthreads();
    }
    stateout[gid] = tState[tidx];
}

__global__
void wholeDiamond(states *statein, states *stateout, int tstep, int dir)
{
    extern __shared__ states tState[];

    int tid = threadIdx.x; // Thread index
    int mid = blockDim.x >> 1; // Half of block size
    int base = blockDim.x + 2; 
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int gidout = (gid + dir*mid) % deqConstants.lastIndex;
    int tidx = tid + 1;
    int tnow = tstep; // read tstep into register.
    int idxes[3];
    for (int k=-1; k<2; k++) idxes[k+1] = tidx + k;

    sweepRead(tState, statein, gid, tid, blockDim.x);

    for (int k=mid; k>0; k--)
	{
		if (tidx < (base-k) && tidx >= k)
		{
        	stepUpdate(tState, idxes, tnow);
		}
		tnow++;
		__syncthreads();
	}

	for (int k=2; k<=mid; k++)
	{
		if (tidx < (base-k) && tidx >= k)
		{
            stepUpdate(tState, idxes, tnow);
		}
		tnow++;
		__syncthreads();
    }
    stateout[gidout] = tState[tidx];
}

Solver::Solver(Equation *eqparam, str scheme, str kernel)
{
    //I'd Prefer to work with a pointer.
    eq = eqparam; //Requires a copy constructor.
    kernelType = kernel;
    schemeType = scheme;
    tstep = TSTEPI;
    bitAlloc = eq->bitSize + 2*eq->stateSize;
    t_eq = NSTEPS * eq->dt;
    twrite = eq->freq - 0.25*eq->dt;
    smem = (eq->tpb + 2) * eq->stateSize;
    

    hipHostAlloc((void **) &hState, bitAlloc, hipHostMallocDefault);
    hipMalloc((void **) &dState, bitAlloc);
    eq->makeInitialCondition(hState);
    hipMemcpyToSymbol(HIP_SYMBOL(deqConstants), &heqConstants, sizeof(equationConstants));
    hipMemcpy(dState, hState, bitAlloc, hipMemcpyHostToDevice);
} 

solver::Solver(Equation *eqparam, std::string scheme, std::string kernel)

~Solver()
{
    hipHostFree(hState);
    hipFree(dState);
    hipDeviceSynchronize();
}


void Solver::classic()
{
    eq->spath += "_Classic_Normal.json";
    eq->tpath += "_Classic_Normal.csv";
    std::cout << "Classic scheme" << std::endl;

    while (t_eq <= eq->tf)
    {
        classicStep <<< eq->bks, eq->tpb >>> (dState, tstep);
    
        tstep++;
        t_eq += (eq->dt * tstep/NSTEPS);

        if (t_eq > twrite)
        {
            this->storeSolution();
            twrite += eq->freq;
        }
    }
}

void Solver::swept()
{
    eq->spath += "_Swept_Normal.json";
    eq->tpath += "_Swept_Normal.csv";
    std::cout << "Swept scheme" << std::endl;

    states *dState2;

    hipMalloc((void **)&dState2, bitAlloc);

    //inline dir = -1, split dir = 1 because passing after calculation.
    upTriangle <<< eq->bks, eq->tpb, smem >>> (dState, dState2, tstep);
    wholeDiamond <<< eq->bks, eq->tpb, smem >>> (dState2, dState, tstep, 1);

    while (t_eq <= eq->tf)
    {
        wholeDiamond <<< eq->bks, eq->tpb, smem >>> (dState, dState2, tstep, -1);
        tstep += eq->height;
        t_eq += (eq->dt * tstep/NSTEPS);

        if (t_eq > twrite)
        {
            downTriangle <<< eq->bks, eq->tpb, smem >>> (dState2, dState, tstep);
            this->storeSolution();
            twrite += eq->freq;
        }
    }   
    hipFree(dState2);
}

void Solver::solveEquation()
{
    cudaTime timer;
    timer.tinit();
    if (!schemeType.compare("S"))
    {
        this->swept();
    }
    else if (!schemeType.compare("C"))
    {
        this->classic();
    }
    else
    {
        std::cerr << "Incorrect or no scheme given! " << std::endl;
    }
    timer.tfinal();
    timed = timer.getLastTime();
}

//Must be accessible from swept and classic.
void Solver::storeSolution()
{
    hipMemcpy(hState, dState, eq->bitSize, hipMemcpyDeviceToHost);
    eq->solutionOutput(hState, t_eq);
}

void Solver::writeFiles()
{
    this->storeSolution();
    double per_ts = timed/(double) tstep;
    timeOut = fopen(eq->tpath.c_str(), "a+");
    fseek(timeOut, 0, SEEK_END);
    int ft = ftell(timeOut);
    if (!ft) fprintf(timeOut, "tpb,nX,time\n");
    fprintf(timeOut, "%d,%d,%.8f\n", eq->tpb, eq->gridSize, per_ts);
    fclose(timeOut);
}